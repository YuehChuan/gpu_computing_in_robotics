#include "hip/hip_runtime.h"
#include "lesson_0.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/count.h>
#include <thrust/copy.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/sort.h>
#include <thrust/sequence.h>
#include <thrust/extrema.h>

__global__ void kernel_cudaWarmUpGPU()
{
	int ind=blockIdx.x*blockDim.x+threadIdx.x;
	ind = ind + 1;
}

hipError_t cudaWarmUpGPU()
{
	kernel_cudaWarmUpGPU<<<1,1>>>();
	hipDeviceSynchronize();
	return hipGetLastError(); 
}

__global__ void kernel_cudaTransformPoints(pcl::PointXYZ *d_point_cloud, int number_of_points, float *d_matrix)
{
	int ind=blockIdx.x*blockDim.x+threadIdx.x;

	if(ind<number_of_points)
	{
		float vSrcVector[3] = {d_point_cloud[ind].x, d_point_cloud[ind].y, d_point_cloud[ind].z};
		float vOut[3];
		vOut[0]=d_matrix[0]*vSrcVector[0]+d_matrix[4]*vSrcVector[1]+d_matrix[8]*vSrcVector[2]+d_matrix[12];
   	 	vOut[1]=d_matrix[1]*vSrcVector[0]+d_matrix[5]*vSrcVector[1]+d_matrix[9]*vSrcVector[2]+d_matrix[13];
    	vOut[2]=d_matrix[2]*vSrcVector[0]+d_matrix[6]*vSrcVector[1]+d_matrix[10]*vSrcVector[2]+d_matrix[14];

		d_point_cloud[ind].x = vOut[0];
		d_point_cloud[ind].y = vOut[1];
		d_point_cloud[ind].z = vOut[2];
	}
}

hipError_t cudaTransformPoints(int threads, pcl::PointXYZ *d_point_cloud, int number_of_points, float *d_matrix)
{
	kernel_cudaTransformPoints<<<number_of_points/threads+1,threads>>>
		(d_point_cloud, number_of_points, d_matrix);

	hipDeviceSynchronize();
	return hipGetLastError();
}

__global__ void kernel_cudaRemovePointsInsideSphere
	(pcl::PointXYZ *d_point_cloud, bool *d_markers, int number_of_points, float sphere_radius)
{
	int ind=blockIdx.x*blockDim.x+threadIdx.x;

	if(ind<number_of_points)
	{
		float x = d_point_cloud[ind].x;
		float y = d_point_cloud[ind].y;
		float z = d_point_cloud[ind].z;
	
		float distance = (x*x + y*y + z*z);
	
		if(distance < sphere_radius * sphere_radius)
		{
			d_markers[ind] = false;
		}else
		{
			d_markers[ind] = true;
		}
	}
}

hipError_t cudaRemovePointsInsideSphere(int threads, pcl::PointXYZ *d_point_cloud, 
		bool *d_markers, int number_of_points, float sphere_radius)
{
	kernel_cudaRemovePointsInsideSphere<<<number_of_points/threads+1,threads>>>
		(d_point_cloud,	d_markers, number_of_points, sphere_radius);
		
	hipDeviceSynchronize();
	return hipGetLastError();
}






