#include "hip/hip_runtime.h"
#include "lesson_6.cuh"

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
//#include <thrust/count.h>
//#include <thrust/copy.h>
//#include <thrust/fill.h>
#include <thrust/sort.h>
//#include <thrust/sequence.h>
#include <thrust/extrema.h>

#include "cuda_SVD.cu"

__global__ void kernel_cudaWarmUpGPU()
{
	int ind=blockIdx.x*blockDim.x+threadIdx.x;
	ind = ind + 1;
}

hipError_t cudaWarmUpGPU()
{
	kernel_cudaWarmUpGPU<<<1,1>>>();
	hipDeviceSynchronize();
	return hipGetLastError();
}

hipError_t cudaCalculateGridParams(pcl::PointNormal* d_point_cloud, int number_of_points,
	float resolution_X, float resolution_Y, float resolution_Z, float bounding_box_extension, gridParameters &out_rgd_params)
{
	hipError_t err = hipGetLastError();

	try
	{
		thrust::device_ptr<pcl::PointNormal> t_cloud(d_point_cloud);
		err = hipGetLastError();
		if(err != ::hipSuccess)return err;
	
		thrust::pair<thrust::device_ptr<pcl::PointNormal>,thrust::device_ptr<pcl::PointNormal> >
		 minmaxX=thrust::minmax_element(t_cloud,t_cloud+number_of_points,compareX());
		err = hipGetLastError();
		if(err != ::hipSuccess)return err;
	
		thrust::pair<thrust::device_ptr<pcl::PointNormal>,thrust::device_ptr<pcl::PointNormal> >
		 minmaxY=thrust::minmax_element(t_cloud,t_cloud+number_of_points,compareY());
		err = hipGetLastError();
		if(err != ::hipSuccess)return err;
	
		thrust::pair<thrust::device_ptr<pcl::PointNormal>,thrust::device_ptr<pcl::PointNormal> >
		 minmaxZ=thrust::minmax_element(t_cloud,t_cloud+number_of_points,compareZ());
		err = hipGetLastError();
		if(err != ::hipSuccess)return err;
		
		pcl::PointNormal minX,maxX,minZ,maxZ,minY,maxY;

		err = hipMemcpy(&minX,minmaxX.first.get(),sizeof(pcl::PointNormal),hipMemcpyDeviceToHost);
		if(err != ::hipSuccess)return err;
		err = hipMemcpy(&maxX,minmaxX.second.get(),sizeof(pcl::PointNormal),hipMemcpyDeviceToHost);
		if(err != ::hipSuccess)return err;
		err = hipMemcpy(&minZ,minmaxZ.first.get(),sizeof(pcl::PointNormal),hipMemcpyDeviceToHost);
		if(err != ::hipSuccess)return err;
		err = hipMemcpy(&maxZ,minmaxZ.second.get(),sizeof(pcl::PointNormal),hipMemcpyDeviceToHost);
		if(err != ::hipSuccess)return err;
		err = hipMemcpy(&minY,minmaxY.first.get(),sizeof(pcl::PointNormal),hipMemcpyDeviceToHost);
		if(err != ::hipSuccess)return err;
		err = hipMemcpy(&maxY,minmaxY.second.get(),sizeof(pcl::PointNormal),hipMemcpyDeviceToHost);
		if(err != ::hipSuccess)return err;
	
		maxX.x += bounding_box_extension;
		minX.x -= bounding_box_extension;

		maxY.y += bounding_box_extension;
		minY.y -= bounding_box_extension;

		maxZ.z += bounding_box_extension;
		minZ.z -= bounding_box_extension;

		int number_of_buckets_X=((maxX.x-minX.x)/resolution_X)+1;
		int number_of_buckets_Y=((maxY.y-minY.y)/resolution_Y)+1;
		int number_of_buckets_Z=((maxZ.z-minZ.z)/resolution_Z)+1;

		out_rgd_params.number_of_buckets_X=number_of_buckets_X;
		out_rgd_params.number_of_buckets_Y=number_of_buckets_Y;
		out_rgd_params.number_of_buckets_Z=number_of_buckets_Z;
		out_rgd_params.number_of_buckets = number_of_buckets_X * number_of_buckets_Y * number_of_buckets_Z;
		
		out_rgd_params.bounding_box_max_X=maxX.x;
		out_rgd_params.bounding_box_min_X=minX.x;
		out_rgd_params.bounding_box_max_Y=maxY.y;
		out_rgd_params.bounding_box_min_Y=minY.y;
		out_rgd_params.bounding_box_max_Z=maxZ.z;
		out_rgd_params.bounding_box_min_Z=minZ.z;
		
		out_rgd_params.resolution_X=resolution_X;
		out_rgd_params.resolution_Y=resolution_Y;
		out_rgd_params.resolution_Z=resolution_Z;
	}
	catch(thrust::system_error &e)
	{
		err = hipGetLastError();
		hipDeviceReset();
		return err;
	}	
	catch(std::bad_alloc &e)
  	{
  	 	err = hipGetLastError();
		hipDeviceReset();
		return err;
  	}	
	return hipGetLastError();
}


__global__ void kernel_initializeIndByKey(hashElement* d_hashTable, int number_of_points)
{
	int ind=blockIdx.x*blockDim.x+threadIdx.x;
	if(ind < number_of_points)
	{
		d_hashTable[ind].index_of_point=ind;
		d_hashTable[ind].index_of_bucket=0;
	}
}

__global__ void kernel_getIndexOfBucketForPoints(pcl::PointNormal* cloud, hashElement* d_hashTable, int number_of_points, gridParameters rgd_params)
{
	int ind=blockIdx.x*blockDim.x+threadIdx.x;
	if(ind < number_of_points)
	{
		int ix=(cloud[ind].x-rgd_params.bounding_box_min_X)/rgd_params.resolution_X;
		int iy=(cloud[ind].y-rgd_params.bounding_box_min_Y)/rgd_params.resolution_Y;
		int iz=(cloud[ind].z-rgd_params.bounding_box_min_Z)/rgd_params.resolution_Z;
		d_hashTable[ind].index_of_bucket=ix*rgd_params.number_of_buckets_Y*rgd_params.number_of_buckets_Z+iy*rgd_params.number_of_buckets_Z+iz;
	}
}

__global__ void kernel_initializeBuckets(bucket* d_buckets, gridParameters rgd_params)
{
	long long int ind=blockIdx.x*blockDim.x+threadIdx.x;
	if(ind < rgd_params.number_of_buckets)
	{
		d_buckets[ind].index_begin=-1;
		d_buckets[ind].index_end=-1;
		d_buckets[ind].number_of_points=0;
	}
}

__global__ void kernel_updateBuckets(hashElement* d_hashTable, bucket* d_buckets,
		gridParameters rgd_params, int number_of_points)
{
	int ind = blockIdx.x*blockDim.x+threadIdx.x;
	if(ind < number_of_points)
	{
		if(ind == 0)
		{
			int index_of_bucket = d_hashTable[ind].index_of_bucket;
			int index_of_bucket_1 = d_hashTable[ind+1].index_of_bucket;

			d_buckets[index_of_bucket].index_begin=ind;
			if(index_of_bucket != index_of_bucket_1)
			{
				d_buckets[index_of_bucket].index_end=ind+1;
				d_buckets[index_of_bucket_1].index_end=ind+1;
			}
		}else if(ind == number_of_points-1)
		{
			d_buckets[d_hashTable[ind].index_of_bucket].index_end=ind+1;
		}else
		{
			int index_of_bucket = d_hashTable[ind].index_of_bucket;
			int index_of_bucket_1 = d_hashTable[ind+1].index_of_bucket;

			if(index_of_bucket != index_of_bucket_1)
			{
				d_buckets[index_of_bucket].index_end=ind+1;
				d_buckets[index_of_bucket_1].index_begin=ind+1;
			}
		}
	}
}

__global__ void kernel_countNumberOfPointsForBuckets(bucket* d_buckets, gridParameters rgd_params)
{
	int ind=blockIdx.x*blockDim.x+threadIdx.x;
	if(ind < rgd_params.number_of_buckets)
	{
		int index_begin = d_buckets[ind].index_begin;
		int index_end = d_buckets[ind].index_end;

		if(index_begin != -1 && index_end !=-1)
		{
			d_buckets[ind].number_of_points = index_end - index_begin;
		}
	}
}

__global__ void kernel_copyKeys(hashElement* d_hashTable_in, hashElement* d_hashTable_out, int number_of_points)
{
	int ind=blockIdx.x*blockDim.x+threadIdx.x;
	if(ind < number_of_points)
	{
		d_hashTable_out[ind] = d_hashTable_in[ind];
	}
}

hipError_t cudaCalculateGrid(int threads, pcl::PointNormal *d_point_cloud, bucket *d_buckets,
		hashElement *d_hashTable, int number_of_points, gridParameters rgd_params)
{
	hipError_t err = hipGetLastError();
	hashElement* d_temp_hashTable;	hipMalloc((void**)&d_temp_hashTable,number_of_points*sizeof(hashElement));
	int blocks=number_of_points/threads + 1;

	kernel_initializeIndByKey<<<blocks,threads>>>(d_temp_hashTable, number_of_points);
	err = hipDeviceSynchronize();	if(err != ::hipSuccess)return err;

	kernel_getIndexOfBucketForPoints<<<blocks,threads>>>(d_point_cloud, d_temp_hashTable, number_of_points, rgd_params);
	err = hipDeviceSynchronize();	if(err != ::hipSuccess)return err;

	try
	{
		thrust::device_ptr<hashElement> t_d_temp_hashTable(d_temp_hashTable);
		thrust::sort(t_d_temp_hashTable,t_d_temp_hashTable+number_of_points,compareHashElements());
	}
	catch(thrust::system_error &e)
	{
		err = hipGetLastError();
		return err;
	}
	catch(std::bad_alloc &e)
	{
		err = hipGetLastError();
		return err;
	}

	kernel_initializeBuckets<<<rgd_params.number_of_buckets/threads+1,threads>>>(d_buckets,rgd_params);
	err = hipDeviceSynchronize();	if(err != ::hipSuccess)return err;

	kernel_updateBuckets<<<blocks,threads>>>(d_temp_hashTable, d_buckets, rgd_params, number_of_points);
	err = hipDeviceSynchronize();	if(err != ::hipSuccess)return err;

	kernel_countNumberOfPointsForBuckets<<<rgd_params.number_of_buckets/threads+1,threads>>>(d_buckets, rgd_params);
	err = hipDeviceSynchronize();	if(err != ::hipSuccess)return err;

	kernel_copyKeys<<<blocks,threads>>>(d_temp_hashTable, d_hashTable, number_of_points);
	err = hipDeviceSynchronize(); if(err != ::hipSuccess)return err;

	err = hipFree(d_temp_hashTable);
	return err;
}

__global__ void kernel_normalvectorcomputation_step1_fast(
		pcl::PointNormal * d_point_cloud,
		hashElement* d_hashTable,
		simple_point3D* d_mean,
		int number_of_points,
		bucket* d_buckets,
		gridParameters rgd_params,
		float search_radius,
		int max_number_considered_in_INNER_bucket,
		int max_number_considered_in_OUTER_bucket)
{
	int index_of_point = blockIdx.x * blockDim.x + threadIdx.x;

	if(index_of_point < number_of_points)
	{
		d_point_cloud[index_of_point].normal_x = 0;
		d_point_cloud[index_of_point].normal_y = 0;
		d_point_cloud[index_of_point].normal_z = 0;
		__syncthreads();

		int index_of_bucket = d_hashTable[index_of_point].index_of_bucket;

		if(index_of_bucket >= 0 && index_of_bucket < rgd_params.number_of_buckets)
		{
			int hashed_index_of_point = d_hashTable[index_of_point].index_of_point;

			if(hashed_index_of_point >= 0 && hashed_index_of_point < number_of_points)
			{
				float x = d_point_cloud[hashed_index_of_point].x;
				float y = d_point_cloud[hashed_index_of_point].y;
				float z = d_point_cloud[hashed_index_of_point].z;

				int ix = index_of_bucket/(rgd_params.number_of_buckets_Y*rgd_params.number_of_buckets_Z);
				int iy = (index_of_bucket%(rgd_params.number_of_buckets_Y*rgd_params.number_of_buckets_Z))/rgd_params.number_of_buckets_Z;
				int iz = (index_of_bucket%(rgd_params.number_of_buckets_Y*rgd_params.number_of_buckets_Z))%rgd_params.number_of_buckets_Z;

				int sx, sy, sz, stx, sty, stz;
				if(ix == 0) sx = 0; else sx = -1;
				if(iy == 0) sy = 0; else sy = -1;
				if(iz == 0) sz = 0; else sz = -1;

				if(ix == rgd_params.number_of_buckets_X - 1)stx = 1; else stx = 2;
				if(iy == rgd_params.number_of_buckets_Y - 1)sty = 1; else sty = 2;
				if(iz == rgd_params.number_of_buckets_Z - 1)stz = 1; else stz = 2;

				int number_of_nearest_neighbours = 0;
				simple_point3D mean;
				mean.x = 0.0f;
				mean.y = 0.0f;
				mean.z = 0.0f;

				float nearest_neighbour_x;
				float nearest_neighbour_y;
				float nearest_neighbour_z;

				for(int i = sx; i < stx; i++)
				{
					for(int j = sy; j < sty; j++)
					{
						for(int k = sz; k < stz; k++)
						{
							int index_of_neighbour_bucket=index_of_bucket+i*rgd_params.number_of_buckets_Y*rgd_params.number_of_buckets_Z+j*rgd_params.number_of_buckets_Z+k;

							if(index_of_neighbour_bucket >= 0 && index_of_neighbour_bucket < rgd_params.number_of_buckets)
							{
								int iter;
								int number_of_points_in_bucket = d_buckets[index_of_neighbour_bucket].number_of_points;
								if(number_of_points_in_bucket <= 0)continue;

								int max_number_considered_in_bucket;
								if(index_of_neighbour_bucket==index_of_bucket)
								{
									max_number_considered_in_bucket = max_number_considered_in_INNER_bucket;
								}else
								{
									max_number_considered_in_bucket = max_number_considered_in_OUTER_bucket;
								}
								if(max_number_considered_in_bucket <= 0)continue;

								if(max_number_considered_in_bucket >= number_of_points_in_bucket)
								{
									iter=1;
								}else
								{
									iter = number_of_points_in_bucket / max_number_considered_in_bucket;
									if(iter <= 0)iter = 1;
								}

								int l_begin = d_buckets[index_of_neighbour_bucket].index_begin;
								int l_end = d_buckets[index_of_neighbour_bucket].index_end;

								for(int l = l_begin; l < l_end; l += iter)
								{
									if(l >= 0 && l < number_of_points)
									{
										int indexNextPointInBucket = d_hashTable[l].index_of_point;
										nearest_neighbour_x = d_point_cloud[indexNextPointInBucket].x;
										nearest_neighbour_y = d_point_cloud[indexNextPointInBucket].y;
										nearest_neighbour_z = d_point_cloud[indexNextPointInBucket].z;

										float dist=sqrtf((x - nearest_neighbour_x)*(x - nearest_neighbour_x)
														+(y - nearest_neighbour_y)*(y - nearest_neighbour_y)
														+(z - nearest_neighbour_z)*(z - nearest_neighbour_z));

										if(dist <= search_radius)
										{
											mean.x += nearest_neighbour_x;
											mean.y += nearest_neighbour_y;
											mean.z += nearest_neighbour_z;
											number_of_nearest_neighbours++;
										}
									}
								}
							}
						}
					}
				}

				if(number_of_nearest_neighbours >= 3)
				{
					d_mean[index_of_point].x = mean.x / number_of_nearest_neighbours;
					d_mean[index_of_point].y = mean.y / number_of_nearest_neighbours;
					d_mean[index_of_point].z = mean.z / number_of_nearest_neighbours;
				}else
				{
					d_mean[index_of_point].x = 0.0f;
					d_mean[index_of_point].y = 0.0f;
					d_mean[index_of_point].z = 0.0f;
				}
			}
		}
	}
}

__global__ void kernel_normalvectorcomputation_step2_fast(
	pcl::PointNormal *d_point_cloud,
	hashElement *d_hashTable,
	simple_point3D *d_mean,
	int number_of_points,
	bucket *d_buckets,
	gridParameters rgd_params,
	float search_radius,
	int max_number_considered_in_INNER_bucket,
	int max_number_considered_in_OUTER_bucket)
{
	int index_of_point = blockIdx.x * blockDim.x + threadIdx.x;
	if(index_of_point < number_of_points)
	{
		int index_of_bucket = d_hashTable[index_of_point].index_of_bucket;
		if(index_of_bucket >= 0 && index_of_bucket < rgd_params.number_of_buckets)
		{
			int hashed_index_of_point = d_hashTable[index_of_point].index_of_point;
			if(hashed_index_of_point >= 0 && hashed_index_of_point < number_of_points)
			{
				simple_point3D mean = d_mean[index_of_point];
				if(mean.x != 0.0f && mean.y != 0.0f && mean.z != 0.0f)
				{
					float x = d_point_cloud[hashed_index_of_point].x;
					float y = d_point_cloud[hashed_index_of_point].y;
					float z = d_point_cloud[hashed_index_of_point].z;

					int ix = index_of_bucket/(rgd_params.number_of_buckets_Y*rgd_params.number_of_buckets_Z);
					int iy = (index_of_bucket%(rgd_params.number_of_buckets_Y*rgd_params.number_of_buckets_Z))/rgd_params.number_of_buckets_Z;
					int iz = (index_of_bucket%(rgd_params.number_of_buckets_Y*rgd_params.number_of_buckets_Z))%rgd_params.number_of_buckets_Z;
					int sx, sy, sz, stx, sty, stz;
					if(ix == 0)sx = 0; else sx = -1;
					if(iy == 0)sy = 0; else sy = -1;
					if(iz == 0)sz = 0; else sz = -1;
					if(ix == rgd_params.number_of_buckets_X - 1)stx = 1; else stx = 2;
					if(iy == rgd_params.number_of_buckets_Y - 1)sty = 1; else sty = 2;
					if(iz == rgd_params.number_of_buckets_Z - 1)stz = 1; else stz = 2;

					int number_of_nearest_neighbours=0;

					double cov[3][3];
					cov[0][0]=cov[0][1]=cov[0][2]=cov[1][0]=cov[1][1]=cov[1][2]=cov[2][0]=cov[2][1]=cov[2][2]=0;

					float nearest_neighbour_x;
					float nearest_neighbour_y;
					float nearest_neighbour_z;

					for(int i = sx; i < stx; i++)
					{
						for(int j = sy; j < sty; j++)
						{
							for(int k = sz; k < stz; k++)
							{
								int index_of_neighbour_bucket=index_of_bucket+i*rgd_params.number_of_buckets_Y*rgd_params.number_of_buckets_Z+j*rgd_params.number_of_buckets_Z+k;
								if(index_of_neighbour_bucket >= 0 && index_of_neighbour_bucket < rgd_params.number_of_buckets)
								{
									int iter;
									int number_of_points_in_bucket = d_buckets[index_of_neighbour_bucket].number_of_points;
									if(number_of_points_in_bucket <= 0)continue;

									int max_number_considered_in_bucket;
									if(index_of_neighbour_bucket==index_of_bucket)
									{
										max_number_considered_in_bucket = max_number_considered_in_INNER_bucket;
									}else
									{
										max_number_considered_in_bucket = max_number_considered_in_OUTER_bucket;
									}
									if(max_number_considered_in_bucket <= 0)continue;


									if(max_number_considered_in_bucket >= number_of_points_in_bucket)
									{
										iter=1;
									}else
									{
										iter = number_of_points_in_bucket / max_number_considered_in_bucket;
										if(iter <= 0)iter = 1;
									}

									int l_begin = d_buckets[index_of_neighbour_bucket].index_begin;
									int l_end = d_buckets[index_of_neighbour_bucket].index_end;

									for(int l = l_begin; l < l_end; l += iter)
									{
										if(l >= 0 && l < number_of_points)
										{
											int indexNextPointInBucket = d_hashTable[l].index_of_point;
											nearest_neighbour_x = d_point_cloud[indexNextPointInBucket].x;
											nearest_neighbour_y = d_point_cloud[indexNextPointInBucket].y;
											nearest_neighbour_z = d_point_cloud[indexNextPointInBucket].z;

											float dist = sqrtf((x - nearest_neighbour_x)*(x - nearest_neighbour_x)
															  +(y - nearest_neighbour_y)*(y - nearest_neighbour_y)
   															  +(z - nearest_neighbour_z)*(z - nearest_neighbour_z));

											if(dist <= search_radius)
											{
												cov[0][0]+=(mean.x - nearest_neighbour_x) * (mean.x - nearest_neighbour_x);
												cov[0][1]+=(mean.x - nearest_neighbour_x) * (mean.y - nearest_neighbour_y);
												cov[0][2]+=(mean.x - nearest_neighbour_x) * (mean.z - nearest_neighbour_z);
												cov[1][0]+=(mean.y - nearest_neighbour_y) * (mean.x - nearest_neighbour_x);
												cov[1][1]+=(mean.y - nearest_neighbour_y) * (mean.y - nearest_neighbour_y);
												cov[1][2]+=(mean.y - nearest_neighbour_y) * (mean.z - nearest_neighbour_z);
												cov[2][0]+=(mean.z - nearest_neighbour_z) * (mean.x - nearest_neighbour_x);
												cov[2][1]+=(mean.z - nearest_neighbour_z) * (mean.y - nearest_neighbour_y);
												cov[2][2]+=(mean.z - nearest_neighbour_z) * (mean.z - nearest_neighbour_z);
												number_of_nearest_neighbours++;
											}
										}
									}
								}
							}
						}
					}

					if(number_of_nearest_neighbours >= 3)
					{
						cov[0][0]/=number_of_nearest_neighbours;
						cov[0][1]/=number_of_nearest_neighbours;
						cov[0][2]/=number_of_nearest_neighbours;
						cov[1][0]/=number_of_nearest_neighbours;
						cov[1][1]/=number_of_nearest_neighbours;
						cov[1][2]/=number_of_nearest_neighbours;
						cov[2][0]/=number_of_nearest_neighbours;
						cov[2][1]/=number_of_nearest_neighbours;
						cov[2][2]/=number_of_nearest_neighbours;

						double U[3][3], V[3][3];
						double SS[9];
						gpuSVD((double *)cov, (double *)U, (double *)SS, (double *)V);
						double _nx = (float)(U[0][1]*U[1][2] - U[0][2]*U[1][1]);
						double _ny = (float)(-(U[0][0]*U[1][2] - U[0][2]*U[1][0] ));
						double _nz = (float)(U[0][0]*U[1][1] - U[0][1]*U[1][0]);

						double lenght = sqrt(_nx*_nx + _ny*_ny + _nz*_nz);
						if(lenght==0)
						{
							d_point_cloud[hashed_index_of_point].normal_x = 0.0f;
							d_point_cloud[hashed_index_of_point].normal_y = 0.0f;
							d_point_cloud[hashed_index_of_point].normal_z = 0.0f;
							d_point_cloud[hashed_index_of_point].curvature = 0.0f;
						}else
						{
							d_point_cloud[hashed_index_of_point].normal_x = _nx/lenght;
							d_point_cloud[hashed_index_of_point].normal_y = _ny/lenght;
							d_point_cloud[hashed_index_of_point].normal_z = _nz/lenght;
							d_point_cloud[hashed_index_of_point].curvature = SS[0]/(SS[0] + SS[4] + SS[8]);
						}
					}
					else
					{
						d_point_cloud[hashed_index_of_point].normal_x = 0.0f;
						d_point_cloud[hashed_index_of_point].normal_y = 0.0f;
						d_point_cloud[hashed_index_of_point].normal_z = 0.0f;
						d_point_cloud[hashed_index_of_point].curvature = 0.0f;
					}
				}
			}
		}
	}
}


hipError_t cudaCalculateNormalVectorsFast(
	int threads,
	pcl::PointNormal * d_point_cloud,
	int number_of_points,
	hashElement* d_hashTable,
	bucket* d_buckets,
	simple_point3D* d_mean,
	gridParameters rgd_params,
	float search_radius,
	int max_number_considered_in_INNER_bucket,
	int max_number_considered_in_OUTER_bucket)
{
	hipError_t err = hipGetLastError();

	int blocks=number_of_points/threads+1;

	kernel_normalvectorcomputation_step1_fast<<<blocks,threads>>>(
		d_point_cloud,
		d_hashTable,
		d_mean,
		number_of_points,
		d_buckets,
		rgd_params,
		search_radius,
		max_number_considered_in_INNER_bucket,
		max_number_considered_in_OUTER_bucket);
	err = hipDeviceSynchronize();
	if(err != ::hipSuccess)return err;

	kernel_normalvectorcomputation_step2_fast<<<blocks,threads>>>(
		d_point_cloud,
		d_hashTable,
		d_mean,
		number_of_points,
		d_buckets,
		rgd_params,
		search_radius,
		max_number_considered_in_INNER_bucket,
		max_number_considered_in_OUTER_bucket);
	err = hipDeviceSynchronize();

	return err;
}

__global__ void  kernel_calculateProjections (
		pcl::PointNormal *d_first_point_cloud,
		int number_of_points_first_point_cloud,
		pcl::PointXYZ *d_second_point_cloud,
		int number_of_points_second_point_cloud,
		hashElement *d_hashTable,
		bucket * d_buckets,
		gridParameters rgd_params,
		int max_number_considered_in_INNER_bucket,
		int max_number_considered_in_OUTER_bucket,
		float projections_search_radius,
		char *d_v_is_projection,
		pcl::PointXYZ *d_second_point_cloud_projections)
{
	int index_of_point_second_point_cloud = blockIdx.x*blockDim.x+threadIdx.x;

	if(index_of_point_second_point_cloud < number_of_points_second_point_cloud)
	{
		bool isok = false;
		float final_projection_distance = 0.0f;
		int   final_nn_index = -1;

		float x_second_point_cloud = d_second_point_cloud[index_of_point_second_point_cloud].x;
		float y_second_point_cloud = d_second_point_cloud[index_of_point_second_point_cloud].y;
		float z_second_point_cloud = d_second_point_cloud[index_of_point_second_point_cloud].z;

		if(x_second_point_cloud < rgd_params.bounding_box_min_X || x_second_point_cloud > rgd_params.bounding_box_max_X)
		{
			d_v_is_projection[index_of_point_second_point_cloud] = false;
			return;
		}
		if(y_second_point_cloud < rgd_params.bounding_box_min_Y || y_second_point_cloud > rgd_params.bounding_box_max_Y)
		{
			d_v_is_projection[index_of_point_second_point_cloud] = false;
			return;
		}
		if(z_second_point_cloud < rgd_params.bounding_box_min_Z || z_second_point_cloud > rgd_params.bounding_box_max_Z)
		{
			d_v_is_projection[index_of_point_second_point_cloud] = false;
			return;
		}

		int ix=(x_second_point_cloud - rgd_params.bounding_box_min_X)/rgd_params.resolution_X;
		int iy=(y_second_point_cloud - rgd_params.bounding_box_min_Y)/rgd_params.resolution_Y;
		int iz=(z_second_point_cloud - rgd_params.bounding_box_min_Z)/rgd_params.resolution_Z;

		int index_bucket = ix*rgd_params.number_of_buckets_Y *
				rgd_params.number_of_buckets_Z + iy * rgd_params.number_of_buckets_Z + iz;

		if(index_bucket >= 0 && index_bucket < rgd_params.number_of_buckets)
		{
			int sx, sy, sz, stx, sty, stz;
			if(ix == 0)sx = 0; else sx = -1;
			if(iy == 0)sy = 0; else sy = -1;
			if(iz == 0)sz = 0; else sz =- 1;

			if(ix == rgd_params.number_of_buckets_X - 1)stx = 1; else stx = 2;
			if(iy == rgd_params.number_of_buckets_Y - 1)sty = 1; else sty = 2;
			if(iz == rgd_params.number_of_buckets_Z - 1)stz = 1; else stz = 2;

			float _distance = 100000000.0f;
			int index_next_bucket;
			int iter;
			int number_of_points_in_bucket;
			int l_begin;
			int l_end;

			for(int i = sx; i < stx; i++)
			{
				for(int j = sy; j < sty; j++)
				{
					for(int k = sz; k < stz; k++)
					{
						index_next_bucket = index_bucket +
								i * rgd_params.number_of_buckets_Y * rgd_params.number_of_buckets_Z +
								j * rgd_params.number_of_buckets_Z + k;
						if(index_next_bucket >= 0 && index_next_bucket < rgd_params.number_of_buckets)
						{
							number_of_points_in_bucket = d_buckets[index_next_bucket].number_of_points;
							if(number_of_points_in_bucket <= 0)continue;

							int max_number_considered_in_bucket;
							if(index_next_bucket == index_bucket)
							{
								max_number_considered_in_bucket = max_number_considered_in_INNER_bucket;
							}else
							{
								max_number_considered_in_bucket = max_number_considered_in_OUTER_bucket;
							}
							if(max_number_considered_in_bucket <= 0)continue;

							if(max_number_considered_in_bucket >= number_of_points_in_bucket)
							{
								iter=1;
							}else
							{
								iter = number_of_points_in_bucket / max_number_considered_in_bucket;
								if(iter <= 0)iter = 1;
							}

							l_begin = d_buckets[index_next_bucket].index_begin;
							l_end = d_buckets[index_next_bucket].index_end;

							for(int l = l_begin; l < l_end; l += iter)
							{
								if(l >= 0 && l < number_of_points_first_point_cloud)
								{
									int hashed_index_of_point = d_hashTable[l].index_of_point;

									float nx_first_point_cloud = d_first_point_cloud[hashed_index_of_point].normal_x;
									float ny_first_point_cloud = d_first_point_cloud[hashed_index_of_point].normal_y;
									float nz_first_point_cloud = d_first_point_cloud[hashed_index_of_point].normal_z;
									float x_first_point_cloud = d_first_point_cloud[hashed_index_of_point].x;
									float y_first_point_cloud = d_first_point_cloud[hashed_index_of_point].y;
									float z_first_point_cloud = d_first_point_cloud[hashed_index_of_point].z;

									float dist  = (x_second_point_cloud - x_first_point_cloud) * (x_second_point_cloud - x_first_point_cloud) +
												  (y_second_point_cloud - y_first_point_cloud) * (y_second_point_cloud - y_first_point_cloud) +
												  (z_second_point_cloud - z_first_point_cloud) * (z_second_point_cloud - z_first_point_cloud);

									float projection_distance = nx_first_point_cloud * x_second_point_cloud +
												ny_first_point_cloud * y_second_point_cloud +
												nz_first_point_cloud * z_second_point_cloud -
												nx_first_point_cloud * x_first_point_cloud -
												ny_first_point_cloud * y_first_point_cloud -
												nz_first_point_cloud * z_first_point_cloud;

									float abs_projection_distance = abs(projection_distance);

									if(dist <= projections_search_radius * projections_search_radius)
									{
										if(abs_projection_distance < _distance)
										{
											isok = true;
											_distance = abs_projection_distance;
											final_projection_distance = projection_distance;
											final_nn_index = hashed_index_of_point;
										}

										//if(dist < _distance)
										//{
										//	isok = true;
										//	nn_index = hashed_index_of_point;
										//	_distance = dist;
										//}
									}
								}
							}
						}
					}
				}
			}
		}

		if(isok)
		{
			d_v_is_projection[index_of_point_second_point_cloud] = 1;
			d_second_point_cloud_projections[index_of_point_second_point_cloud].x =
					d_second_point_cloud[index_of_point_second_point_cloud].x -
					d_first_point_cloud[final_nn_index].normal_x * final_projection_distance;

			d_second_point_cloud_projections[index_of_point_second_point_cloud].y =
					d_second_point_cloud[index_of_point_second_point_cloud].y -
					d_first_point_cloud[final_nn_index].normal_y * final_projection_distance;

			d_second_point_cloud_projections[index_of_point_second_point_cloud].z =
					d_second_point_cloud[index_of_point_second_point_cloud].z -
					d_first_point_cloud[final_nn_index].normal_z * final_projection_distance;

		}else
		{
			d_v_is_projection[index_of_point_second_point_cloud] = 0;
			d_second_point_cloud_projections[index_of_point_second_point_cloud].x = 0.0f;
			d_second_point_cloud_projections[index_of_point_second_point_cloud].y = 0.0f;
			d_second_point_cloud_projections[index_of_point_second_point_cloud].z = 0.0f;
		}
	}
}

hipError_t cudaCalculateProjections(
		int threads,
		pcl::PointNormal *d_first_point_cloud,
		int number_of_points_first_point_cloud,
		pcl::PointXYZ *d_second_point_cloud,
		int number_of_points_second_point_cloud,
		hashElement *d_hashTable,
		bucket * d_buckets,
		gridParameters rgd_params,
		int max_number_considered_in_INNER_bucket,
		int max_number_considered_in_OUTER_bucket,
		float projections_search_radius,
		char *d_v_is_projection,
		pcl::PointXYZ *d_second_point_cloud_projections)
{
	hipError_t err = hipGetLastError();
	if(err != ::hipSuccess)return err;

	int blocks=number_of_points_second_point_cloud/threads+1;

	kernel_calculateProjections<<<blocks, threads>>> (
			d_first_point_cloud,
			number_of_points_first_point_cloud,
			d_second_point_cloud,
			number_of_points_second_point_cloud,
			d_hashTable,
			d_buckets,
			rgd_params,
			max_number_considered_in_INNER_bucket,
			max_number_considered_in_OUTER_bucket,
			projections_search_radius,
			d_v_is_projection,
			d_second_point_cloud_projections);
	err = hipDeviceSynchronize();

	return err;
}

__global__ void kernel_cudaTransformPoints(pcl::PointXYZ *d_point_cloud, int number_of_points, float *d_matrix)
{
	int ind=blockIdx.x*blockDim.x+threadIdx.x;

	if(ind<number_of_points)
	{
		float vSrcVector[3] = {d_point_cloud[ind].x, d_point_cloud[ind].y, d_point_cloud[ind].z};
		float vOut[3];
		vOut[0]=d_matrix[0]*vSrcVector[0]+d_matrix[4]*vSrcVector[1]+d_matrix[8]*vSrcVector[2]+d_matrix[12];
   	 	vOut[1]=d_matrix[1]*vSrcVector[0]+d_matrix[5]*vSrcVector[1]+d_matrix[9]*vSrcVector[2]+d_matrix[13];
    	vOut[2]=d_matrix[2]*vSrcVector[0]+d_matrix[6]*vSrcVector[1]+d_matrix[10]*vSrcVector[2]+d_matrix[14];

		d_point_cloud[ind].x = vOut[0];
		d_point_cloud[ind].y = vOut[1];
		d_point_cloud[ind].z = vOut[2];
	}
}

hipError_t cudaTransformPoints(int threads, pcl::PointXYZ *d_point_cloud, int number_of_points, float *d_matrix)
{
	kernel_cudaTransformPoints<<<number_of_points/threads+1,threads>>>
		(d_point_cloud, number_of_points, d_matrix);

	hipDeviceSynchronize();
	return hipGetLastError();
}

