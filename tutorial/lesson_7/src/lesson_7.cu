#include "hip/hip_runtime.h"
#include "lesson_7.cuh"

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
//#include <thrust/count.h>
//#include <thrust/copy.h>
//#include <thrust/fill.h>
#include <thrust/sort.h>
//#include <thrust/sequence.h>
#include <thrust/extrema.h>

#include "cuda_SVD.cu"

__global__ void kernel_cudaWarmUpGPU()
{
	int ind=blockIdx.x*blockDim.x+threadIdx.x;
	ind = ind + 1;
}

hipError_t cudaWarmUpGPU()
{
	kernel_cudaWarmUpGPU<<<1,1>>>();
	hipDeviceSynchronize();
	return hipGetLastError();
}

hipError_t cudaCalculateGridParams(VelodyneVLP16::PointXYZNL* d_point_cloud, int number_of_points,
	float resolution_X, float resolution_Y, float resolution_Z, gridParameters &out_rgd_params)
{
	hipError_t err = hipGetLastError();

	try
	{
		thrust::device_ptr<VelodyneVLP16::PointXYZNL> t_cloud(d_point_cloud);
		err = hipGetLastError();
		if(err != ::hipSuccess)return err;
	
		thrust::pair<thrust::device_ptr<VelodyneVLP16::PointXYZNL>,thrust::device_ptr<VelodyneVLP16::PointXYZNL> >
		 minmaxX=thrust::minmax_element(t_cloud,t_cloud+number_of_points,compareX());
		err = hipGetLastError();
		if(err != ::hipSuccess)return err;
	
		thrust::pair<thrust::device_ptr<VelodyneVLP16::PointXYZNL>,thrust::device_ptr<VelodyneVLP16::PointXYZNL> >
		 minmaxY=thrust::minmax_element(t_cloud,t_cloud+number_of_points,compareY());
		err = hipGetLastError();
		if(err != ::hipSuccess)return err;
	
		thrust::pair<thrust::device_ptr<VelodyneVLP16::PointXYZNL>,thrust::device_ptr<VelodyneVLP16::PointXYZNL> >
		 minmaxZ=thrust::minmax_element(t_cloud,t_cloud+number_of_points,compareZ());
		err = hipGetLastError();
		if(err != ::hipSuccess)return err;
		
		VelodyneVLP16::PointXYZNL minX,maxX,minZ,maxZ,minY,maxY;

		err = hipMemcpy(&minX,minmaxX.first.get(),sizeof(VelodyneVLP16::PointXYZNL),hipMemcpyDeviceToHost);
		if(err != ::hipSuccess)return err;
		err = hipMemcpy(&maxX,minmaxX.second.get(),sizeof(VelodyneVLP16::PointXYZNL),hipMemcpyDeviceToHost);
		if(err != ::hipSuccess)return err;
		err = hipMemcpy(&minZ,minmaxZ.first.get(),sizeof(VelodyneVLP16::PointXYZNL),hipMemcpyDeviceToHost);
		if(err != ::hipSuccess)return err;
		err = hipMemcpy(&maxZ,minmaxZ.second.get(),sizeof(VelodyneVLP16::PointXYZNL),hipMemcpyDeviceToHost);
		if(err != ::hipSuccess)return err;
		err = hipMemcpy(&minY,minmaxY.first.get(),sizeof(VelodyneVLP16::PointXYZNL),hipMemcpyDeviceToHost);
		if(err != ::hipSuccess)return err;
		err = hipMemcpy(&maxY,minmaxY.second.get(),sizeof(VelodyneVLP16::PointXYZNL),hipMemcpyDeviceToHost);
		if(err != ::hipSuccess)return err;
	
		int number_of_buckets_X=((maxX.x-minX.x)/resolution_X)+1;
		int number_of_buckets_Y=((maxY.y-minY.y)/resolution_Y)+1;
		int number_of_buckets_Z=((maxZ.z-minZ.z)/resolution_Z)+1;

		out_rgd_params.number_of_buckets_X=number_of_buckets_X;
		out_rgd_params.number_of_buckets_Y=number_of_buckets_Y;
		out_rgd_params.number_of_buckets_Z=number_of_buckets_Z;
		out_rgd_params.number_of_buckets = number_of_buckets_X * number_of_buckets_Y * number_of_buckets_Z;
		
		out_rgd_params.bounding_box_max_X=maxX.x;
		out_rgd_params.bounding_box_min_X=minX.x;
		out_rgd_params.bounding_box_max_Y=maxY.y;
		out_rgd_params.bounding_box_min_Y=minY.y;
		out_rgd_params.bounding_box_max_Z=maxZ.z;
		out_rgd_params.bounding_box_min_Z=minZ.z;
		
		out_rgd_params.resolution_X=resolution_X;
		out_rgd_params.resolution_Y=resolution_Y;
		out_rgd_params.resolution_Z=resolution_Z;

	}
	catch(thrust::system_error &e)
	{
		err = hipGetLastError();
		hipDeviceReset();
		return err;
	}	
	catch(std::bad_alloc &e)
  	{
  	 	err = hipGetLastError();
		hipDeviceReset();
		return err;
  	}	
	return hipGetLastError();
}

__global__ void kernel_initializeIndByKey(hashElement* d_hashTable, int number_of_points)
{
	int ind=blockIdx.x*blockDim.x+threadIdx.x;
	if(ind < number_of_points)
	{
		d_hashTable[ind].index_of_point=ind;
		d_hashTable[ind].index_of_bucket=0;
	}
}

__global__ void kernel_getIndexOfBucketForPoints(VelodyneVLP16::PointXYZNL* d_point_cloud,
		hashElement* d_hashTable, int number_of_points, gridParameters rgd_params)
{
	int ind=blockIdx.x*blockDim.x+threadIdx.x;
	if(ind<number_of_points)
	{
		int ix=(d_point_cloud[ind].x-rgd_params.bounding_box_min_X)/rgd_params.resolution_X;
		int iy=(d_point_cloud[ind].y-rgd_params.bounding_box_min_Y)/rgd_params.resolution_Y;
		int iz=(d_point_cloud[ind].z-rgd_params.bounding_box_min_Z)/rgd_params.resolution_Z;
		d_hashTable[ind].index_of_bucket=ix*rgd_params.number_of_buckets_Y*rgd_params.number_of_buckets_Z+iy*rgd_params.number_of_buckets_Z+iz;
	}
}

__global__ void kernel_initializeBuckets(bucket* d_buckets, gridParameters rgd_params)
{
	long long int ind=blockIdx.x*blockDim.x+threadIdx.x;
	if(ind < rgd_params.number_of_buckets)
	{
		d_buckets[ind].index_begin=-1;
		d_buckets[ind].index_end=-1;
		d_buckets[ind].number_of_points=0;
	}
}

__global__ void kernel_updateBuckets(hashElement* d_hashTable, bucket* d_buckets,
		gridParameters rgd_params, int number_of_points)
{
	int ind = blockIdx.x*blockDim.x+threadIdx.x;
	if(ind < number_of_points)
	{
		if(ind == 0)
		{
			int index_of_bucket = d_hashTable[ind].index_of_bucket;
			int index_of_bucket_1 = d_hashTable[ind+1].index_of_bucket;

			d_buckets[index_of_bucket].index_begin=ind;
			if(index_of_bucket != index_of_bucket_1)
			{
				d_buckets[index_of_bucket].index_end=ind+1;
				d_buckets[index_of_bucket_1].index_end=ind+1;
			}
		}else if(ind == number_of_points-1)
		{
			d_buckets[d_hashTable[ind].index_of_bucket].index_end=ind+1;
		}else
		{
			int index_of_bucket = d_hashTable[ind].index_of_bucket;
			int index_of_bucket_1 = d_hashTable[ind+1].index_of_bucket;

			if(index_of_bucket != index_of_bucket_1)
			{
				d_buckets[index_of_bucket].index_end=ind+1;
				d_buckets[index_of_bucket_1].index_begin=ind+1;
			}
		}
	}
}

__global__ void kernel_countNumberOfPointsForBuckets(bucket* d_buckets, gridParameters rgd_params)
{
	int ind=blockIdx.x*blockDim.x+threadIdx.x;
	if(ind < rgd_params.number_of_buckets)
	{
		int index_begin = d_buckets[ind].index_begin;
		int index_end = d_buckets[ind].index_end;

		if(index_begin != -1 && index_end !=-1)
		{
			d_buckets[ind].number_of_points = index_end - index_begin;
		}
	}
}

__global__ void kernel_copyKeys(hashElement* d_hashTable_in, hashElement* d_hashTable_out, int number_of_points)
{
	int ind=blockIdx.x*blockDim.x+threadIdx.x;
	if(ind < number_of_points)
	{
		d_hashTable_out[ind] = d_hashTable_in[ind];
	}
}

hipError_t cudaCalculateGrid(int threads, VelodyneVLP16::PointXYZNL *d_point_cloud, bucket *d_buckets,
		hashElement *d_hashTable, int number_of_points, gridParameters rgd_params)
{
	hipError_t err = hipGetLastError();
	hashElement* d_temp_hashTable;	hipMalloc((void**)&d_temp_hashTable,number_of_points*sizeof(hashElement));
	int blocks=number_of_points/threads + 1;
	
	kernel_initializeIndByKey<<<blocks,threads>>>(d_temp_hashTable, number_of_points);
	err = hipDeviceSynchronize();	if(err != ::hipSuccess)return err;
	
	kernel_getIndexOfBucketForPoints<<<blocks,threads>>>(d_point_cloud, d_temp_hashTable, number_of_points, rgd_params);
	err = hipDeviceSynchronize();	if(err != ::hipSuccess)return err;

	try
	{
		thrust::device_ptr<hashElement> t_d_temp_hashTable(d_temp_hashTable);
		thrust::sort(t_d_temp_hashTable,t_d_temp_hashTable+number_of_points,compareHashElements());
	}
	catch(thrust::system_error &e)	
	{
		err = hipGetLastError(); 
		return err;
	}	
	catch(std::bad_alloc &e)
	{
		err = hipGetLastError(); 
		return err;
	}	

	kernel_initializeBuckets<<<rgd_params.number_of_buckets/threads+1,threads>>>(d_buckets,rgd_params);
	err = hipDeviceSynchronize();	if(err != ::hipSuccess)return err;
	
	kernel_updateBuckets<<<blocks,threads>>>(d_temp_hashTable, d_buckets, rgd_params, number_of_points);
	err = hipDeviceSynchronize();	if(err != ::hipSuccess)return err;
	
	kernel_countNumberOfPointsForBuckets<<<rgd_params.number_of_buckets/threads+1,threads>>>(d_buckets, rgd_params);
	err = hipDeviceSynchronize();	if(err != ::hipSuccess)return err;
	
	kernel_copyKeys<<<blocks,threads>>>(d_temp_hashTable, d_hashTable, number_of_points);
	err = hipDeviceSynchronize(); if(err != ::hipSuccess)return err;
	
	err = hipFree(d_temp_hashTable);
	return err;
}

/////////////////////////////////computeNormalVectors/////////////////////////////////////
__global__ void kernel_normalvectorcomputation_step1_fast(
		VelodyneVLP16::PointXYZNL * d_point_cloud,
		hashElement* d_hashTable,
		simple_point3D* d_mean,
		int number_of_points,
		bucket* d_buckets, 
		gridParameters rgd_params,
		float search_radius, 
		int max_number_considered_in_INNER_bucket, 
		int max_number_considered_in_OUTER_bucket)
{
	int index_of_point = blockIdx.x * blockDim.x + threadIdx.x;

	if(index_of_point < number_of_points)
	{
		d_point_cloud[index_of_point].normal_x = 0;
		d_point_cloud[index_of_point].normal_y = 0;
		d_point_cloud[index_of_point].normal_z = 0;
		__syncthreads();

		int index_of_bucket = d_hashTable[index_of_point].index_of_bucket;
		
		if(index_of_bucket >= 0 && index_of_bucket < rgd_params.number_of_buckets)
		{
			int hashed_index_of_point = d_hashTable[index_of_point].index_of_point;

			if(hashed_index_of_point >= 0 && hashed_index_of_point < number_of_points)
			{
				float x = d_point_cloud[hashed_index_of_point].x;
				float y = d_point_cloud[hashed_index_of_point].y;
				float z = d_point_cloud[hashed_index_of_point].z;

				int ix = index_of_bucket/(rgd_params.number_of_buckets_Y*rgd_params.number_of_buckets_Z);
				int iy = (index_of_bucket%(rgd_params.number_of_buckets_Y*rgd_params.number_of_buckets_Z))/rgd_params.number_of_buckets_Z;
				int iz = (index_of_bucket%(rgd_params.number_of_buckets_Y*rgd_params.number_of_buckets_Z))%rgd_params.number_of_buckets_Z;

				int sx, sy, sz, stx, sty, stz;
				if(ix == 0) sx = 0; else sx = -1;
				if(iy == 0) sy = 0; else sy = -1;
				if(iz == 0) sz = 0; else sz = -1;

				if(ix == rgd_params.number_of_buckets_X - 1)stx = 1; else stx = 2;
				if(iy == rgd_params.number_of_buckets_Y - 1)sty = 1; else sty = 2;
				if(iz == rgd_params.number_of_buckets_Z - 1)stz = 1; else stz = 2;

				int number_of_nearest_neighbours = 0;
				simple_point3D mean;
				mean.x = 0.0f;
				mean.y = 0.0f;
				mean.z = 0.0f;

				float nearest_neighbour_x;
				float nearest_neighbour_y;
				float nearest_neighbour_z;
			
				for(int i = sx; i < stx; i++)
				{
					for(int j = sy; j < sty; j++)
					{
						for(int k = sz; k < stz; k++)
						{
							int index_of_neighbour_bucket=index_of_bucket+i*rgd_params.number_of_buckets_Y*rgd_params.number_of_buckets_Z+j*rgd_params.number_of_buckets_Z+k;

							if(index_of_neighbour_bucket >= 0 && index_of_neighbour_bucket < rgd_params.number_of_buckets)
							{
								int iter;
								int number_of_points_in_bucket = d_buckets[index_of_neighbour_bucket].number_of_points;
								if(number_of_points_in_bucket <= 0)continue;

								int max_number_considered_in_bucket;
								if(index_of_neighbour_bucket==index_of_bucket)
								{
									max_number_considered_in_bucket = max_number_considered_in_INNER_bucket;
								}else
								{
									max_number_considered_in_bucket = max_number_considered_in_OUTER_bucket;
								}
								if(max_number_considered_in_bucket <= 0)continue;

								if(max_number_considered_in_bucket >= number_of_points_in_bucket)
								{
									iter=1;
								}else
								{
									iter = number_of_points_in_bucket / max_number_considered_in_bucket;
									if(iter <= 0)iter = 1;
								}

								int l_begin = d_buckets[index_of_neighbour_bucket].index_begin;
								int l_end = d_buckets[index_of_neighbour_bucket].index_end;

								for(int l = l_begin; l < l_end; l += iter)
								{
									if(l >= 0 && l < number_of_points)
									{
										int indexNextPointInBucket = d_hashTable[l].index_of_point;
										nearest_neighbour_x = d_point_cloud[indexNextPointInBucket].x;
										nearest_neighbour_y = d_point_cloud[indexNextPointInBucket].y;
										nearest_neighbour_z = d_point_cloud[indexNextPointInBucket].z;

										float dist=sqrtf((x - nearest_neighbour_x)*(x - nearest_neighbour_x)
														+(y - nearest_neighbour_y)*(y - nearest_neighbour_y)
														+(z - nearest_neighbour_z)*(z - nearest_neighbour_z));

										if(dist <= search_radius)
										{
											mean.x += nearest_neighbour_x;
											mean.y += nearest_neighbour_y;
											mean.z += nearest_neighbour_z;
											number_of_nearest_neighbours++;
										}
									}
								}
							}
						}
					}
				}
				
				if(number_of_nearest_neighbours >= 3)
				{
					d_mean[index_of_point].x = mean.x / number_of_nearest_neighbours;
					d_mean[index_of_point].y = mean.y / number_of_nearest_neighbours;
					d_mean[index_of_point].z = mean.z / number_of_nearest_neighbours;
				}else
				{
					d_mean[index_of_point].x = 0.0f;
					d_mean[index_of_point].y = 0.0f;
					d_mean[index_of_point].z = 0.0f;
				}
			}
		}
	}
}

__global__ void kernel_normalvectorcomputation_step2_fast_with_classification(
	VelodyneVLP16::PointXYZNL *d_point_cloud,
	hashElement *d_hashTable,
	simple_point3D *d_mean,
	int number_of_points,
	bucket *d_buckets,
	gridParameters rgd_params,
	float search_radius, 
	int max_number_considered_in_INNER_bucket, 
	int max_number_considered_in_OUTER_bucket,
	float curvature_threshold,
	int number_of_points_needed_for_plane_threshold)
{
	int index_of_point = blockIdx.x * blockDim.x + threadIdx.x;
	if(index_of_point < number_of_points)
	{
		int index_of_bucket = d_hashTable[index_of_point].index_of_bucket;
		if(index_of_bucket >= 0 && index_of_bucket < rgd_params.number_of_buckets)
		{
			int hashed_index_of_point = d_hashTable[index_of_point].index_of_point;
			if(hashed_index_of_point >= 0 && hashed_index_of_point < number_of_points)
			{
				d_point_cloud[hashed_index_of_point].label = 1;

				simple_point3D mean = d_mean[index_of_point];
				if(mean.x != 0.0f && mean.y != 0.0f && mean.z != 0.0f)
				{
					float x = d_point_cloud[hashed_index_of_point].x;
					float y = d_point_cloud[hashed_index_of_point].y;
					float z = d_point_cloud[hashed_index_of_point].z;

					int ix = index_of_bucket/(rgd_params.number_of_buckets_Y*rgd_params.number_of_buckets_Z);
					int iy = (index_of_bucket%(rgd_params.number_of_buckets_Y*rgd_params.number_of_buckets_Z))/rgd_params.number_of_buckets_Z;
					int iz = (index_of_bucket%(rgd_params.number_of_buckets_Y*rgd_params.number_of_buckets_Z))%rgd_params.number_of_buckets_Z;
					int sx, sy, sz, stx, sty, stz;
					if(ix == 0)sx = 0; else sx = -1;
					if(iy == 0)sy = 0; else sy = -1;
					if(iz == 0)sz = 0; else sz = -1;
					if(ix == rgd_params.number_of_buckets_X - 1)stx = 1; else stx = 2;
					if(iy == rgd_params.number_of_buckets_Y - 1)sty = 1; else sty = 2;
					if(iz == rgd_params.number_of_buckets_Z - 1)stz = 1; else stz = 2;

					int number_of_nearest_neighbours=0;

					double cov[3][3];
					cov[0][0]=cov[0][1]=cov[0][2]=cov[1][0]=cov[1][1]=cov[1][2]=cov[2][0]=cov[2][1]=cov[2][2]=0;

					float nearest_neighbour_x;
					float nearest_neighbour_y;
					float nearest_neighbour_z;
					
					for(int i = sx; i < stx; i++)
					{
						for(int j = sy; j < sty; j++)
						{
							for(int k = sz; k < stz; k++)
							{
								int index_of_neighbour_bucket=index_of_bucket+i*rgd_params.number_of_buckets_Y*rgd_params.number_of_buckets_Z+j*rgd_params.number_of_buckets_Z+k;
								if(index_of_neighbour_bucket >= 0 && index_of_neighbour_bucket < rgd_params.number_of_buckets)
								{
									int iter;
									int number_of_points_in_bucket = d_buckets[index_of_neighbour_bucket].number_of_points;
									if(number_of_points_in_bucket <= 0)continue;

									int max_number_considered_in_bucket;
									if(index_of_neighbour_bucket==index_of_bucket)
									{
										max_number_considered_in_bucket = max_number_considered_in_INNER_bucket;
									}else
									{
										max_number_considered_in_bucket = max_number_considered_in_OUTER_bucket;
									}
									if(max_number_considered_in_bucket <= 0)continue;


									if(max_number_considered_in_bucket >= number_of_points_in_bucket)
									{
										iter=1;
									}else
									{
										iter = number_of_points_in_bucket / max_number_considered_in_bucket;
										if(iter <= 0)iter = 1;
									}

									int l_begin = d_buckets[index_of_neighbour_bucket].index_begin;
									int l_end = d_buckets[index_of_neighbour_bucket].index_end;

									for(int l = l_begin; l < l_end; l += iter)
									{
										if(l >= 0 && l < number_of_points)
										{
											int indexNextPointInBucket = d_hashTable[l].index_of_point;
											nearest_neighbour_x = d_point_cloud[indexNextPointInBucket].x;
											nearest_neighbour_y = d_point_cloud[indexNextPointInBucket].y;
											nearest_neighbour_z = d_point_cloud[indexNextPointInBucket].z;

											float dist = sqrtf((x - nearest_neighbour_x)*(x - nearest_neighbour_x)
															  +(y - nearest_neighbour_y)*(y - nearest_neighbour_y)
   															  +(z - nearest_neighbour_z)*(z - nearest_neighbour_z));

											if(dist <= search_radius)
											{
												cov[0][0]+=(mean.x - nearest_neighbour_x) * (mean.x - nearest_neighbour_x);
												cov[0][1]+=(mean.x - nearest_neighbour_x) * (mean.y - nearest_neighbour_y);
												cov[0][2]+=(mean.x - nearest_neighbour_x) * (mean.z - nearest_neighbour_z);
												cov[1][0]+=(mean.y - nearest_neighbour_y) * (mean.x - nearest_neighbour_x);
												cov[1][1]+=(mean.y - nearest_neighbour_y) * (mean.y - nearest_neighbour_y);
												cov[1][2]+=(mean.y - nearest_neighbour_y) * (mean.z - nearest_neighbour_z);
												cov[2][0]+=(mean.z - nearest_neighbour_z) * (mean.x - nearest_neighbour_x);
												cov[2][1]+=(mean.z - nearest_neighbour_z) * (mean.y - nearest_neighbour_y);
												cov[2][2]+=(mean.z - nearest_neighbour_z) * (mean.z - nearest_neighbour_z);
												number_of_nearest_neighbours++;
											}
										}
									}
								}
							}
						}
					}

					if(number_of_nearest_neighbours >= number_of_points_needed_for_plane_threshold)
					{
						cov[0][0]/=number_of_nearest_neighbours;
						cov[0][1]/=number_of_nearest_neighbours;
						cov[0][2]/=number_of_nearest_neighbours;
						cov[1][0]/=number_of_nearest_neighbours;
						cov[1][1]/=number_of_nearest_neighbours;
						cov[1][2]/=number_of_nearest_neighbours;
						cov[2][0]/=number_of_nearest_neighbours;
						cov[2][1]/=number_of_nearest_neighbours;
						cov[2][2]/=number_of_nearest_neighbours;

						double U[3][3], V[3][3];
						double SS[9];
						gpuSVD((double *)cov, (double *)U, (double *)SS, (double *)V);
						double _nx = (float)(U[0][1]*U[1][2] - U[0][2]*U[1][1]);
						double _ny = (float)(-(U[0][0]*U[1][2] - U[0][2]*U[1][0] ));
						double _nz = (float)(U[0][0]*U[1][1] - U[0][1]*U[1][0]);

						double lenght = sqrt(_nx*_nx + _ny*_ny + _nz*_nz);
						if(lenght==0)
						{
							d_point_cloud[hashed_index_of_point].normal_x = 0.0f;
							d_point_cloud[hashed_index_of_point].normal_y = 0.0f;
							d_point_cloud[hashed_index_of_point].normal_z = 0.0f;
						}else
						{
							d_point_cloud[hashed_index_of_point].normal_x = _nx/lenght;
							d_point_cloud[hashed_index_of_point].normal_y = _ny/lenght;
							d_point_cloud[hashed_index_of_point].normal_z = _nz/lenght;
							if( (SS[4]/SS[8]) > curvature_threshold)
							{
								d_point_cloud[hashed_index_of_point].label = 0;
							}
						}
					}
					else
					{
						d_point_cloud[hashed_index_of_point].normal_x = 0.0f;
						d_point_cloud[hashed_index_of_point].normal_y = 0.0f;
						d_point_cloud[hashed_index_of_point].normal_z = 0.0f;
					}
				}
			}
		}
	}
}

hipError_t cudaSemanticLabelingPlaneEdges(
	int threads,
	VelodyneVLP16::PointXYZNL * d_point_cloud,
	int number_of_points,
	hashElement* d_hashTable,
	bucket* d_buckets,
	simple_point3D* d_mean,
	gridParameters rgd_params,
	float search_radius,
	int max_number_considered_in_INNER_bucket,
	int max_number_considered_in_OUTER_bucket,
	float curvature_threshold,
	int number_of_points_needed_for_plane_threshold)
{
	hipError_t err = hipGetLastError();

	int blocks=number_of_points/threads+1;

	kernel_normalvectorcomputation_step1_fast<<<blocks,threads>>>(
		d_point_cloud, 
		d_hashTable,
		d_mean,
		number_of_points,
		d_buckets, 
		rgd_params,
		search_radius, 
		max_number_considered_in_INNER_bucket, 
		max_number_considered_in_OUTER_bucket);
	err = hipDeviceSynchronize();
	if(err != ::hipSuccess)return err;

	kernel_normalvectorcomputation_step2_fast_with_classification<<<blocks,threads>>>(
		d_point_cloud, 
		d_hashTable,
		d_mean,
		number_of_points,
		d_buckets,
		rgd_params,
		search_radius, 
		max_number_considered_in_INNER_bucket, 
		max_number_considered_in_OUTER_bucket,
		curvature_threshold,
		number_of_points_needed_for_plane_threshold);
	err = hipDeviceSynchronize();
	
	return err;
}

__global__ void kernel_semanticLabelingFloorCeiling(
			int threads,
			VelodyneVLP16::PointXYZNL * d_point_cloud,
			int number_of_points,
			float ground_Z_coordinate_threshold)
{
	int index_of_point = blockIdx.x * blockDim.x + threadIdx.x;
	if(index_of_point < number_of_points)
	{
		if(d_point_cloud[index_of_point].label == 0)
		{
			if(d_point_cloud[index_of_point].normal_z > 0.7 || d_point_cloud[index_of_point].normal_z < -0.7)
			{
				if(d_point_cloud[index_of_point].z < ground_Z_coordinate_threshold)
				{
					d_point_cloud[index_of_point].label = 3;
				}else
				{
					d_point_cloud[index_of_point].label = 2;
				}
			}
		}
	}
}

hipError_t cudaSemanticLabelingFloorCeiling(
		int threads,
		VelodyneVLP16::PointXYZNL * d_point_cloud,
		int number_of_points,
		float ground_Z_coordinate_threshold)
{
	hipError_t err = hipGetLastError();
	int blocks=number_of_points/threads+1;

	kernel_semanticLabelingFloorCeiling<<<blocks,threads>>>(
			threads,
			d_point_cloud,
			number_of_points,
			ground_Z_coordinate_threshold);

	err = hipDeviceSynchronize();
	return err;
}

