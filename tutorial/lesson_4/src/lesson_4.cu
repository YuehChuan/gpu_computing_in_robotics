#include "hip/hip_runtime.h"
#include "lesson_4.cuh"

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
//#include <thrust/count.h>
//#include <thrust/copy.h>
//#include <thrust/fill.h>
#include <thrust/sort.h>
//#include <thrust/sequence.h>
#include <thrust/extrema.h>

__global__ void kernel_cudaWarmUpGPU()
{
	int ind=blockIdx.x*blockDim.x+threadIdx.x;
	ind = ind + 1;
}

hipError_t cudaWarmUpGPU()
{
	kernel_cudaWarmUpGPU<<<1,1>>>();
	hipDeviceSynchronize();
	return hipGetLastError();
}

hipError_t cudaCalculateGridParams(pcl::PointXYZ* d_point_cloud, int number_of_points,
	float resolution_X, float resolution_Y, float resolution_Z, gridParameters &out_rgd_params)
{
	hipError_t err = hipGetLastError();

	try
	{
		thrust::device_ptr<pcl::PointXYZ> t_cloud(d_point_cloud);
		err = hipGetLastError();
		if(err != ::hipSuccess)return err;
	
		thrust::pair<thrust::device_ptr<pcl::PointXYZ>,thrust::device_ptr<pcl::PointXYZ> >
		 minmaxX=thrust::minmax_element(t_cloud,t_cloud+number_of_points,compareX());
		err = hipGetLastError();
		if(err != ::hipSuccess)return err;
	
		thrust::pair<thrust::device_ptr<pcl::PointXYZ>,thrust::device_ptr<pcl::PointXYZ> >
		 minmaxY=thrust::minmax_element(t_cloud,t_cloud+number_of_points,compareY());
		err = hipGetLastError();
		if(err != ::hipSuccess)return err;
	
		thrust::pair<thrust::device_ptr<pcl::PointXYZ>,thrust::device_ptr<pcl::PointXYZ> >
		 minmaxZ=thrust::minmax_element(t_cloud,t_cloud+number_of_points,compareZ());
		err = hipGetLastError();
		if(err != ::hipSuccess)return err;
		
		pcl::PointXYZ minX,maxX,minZ,maxZ,minY,maxY;

		err = hipMemcpy(&minX,minmaxX.first.get(),sizeof(pcl::PointXYZ),hipMemcpyDeviceToHost);
		if(err != ::hipSuccess)return err;
		err = hipMemcpy(&maxX,minmaxX.second.get(),sizeof(pcl::PointXYZ),hipMemcpyDeviceToHost);
		if(err != ::hipSuccess)return err;
		err = hipMemcpy(&minZ,minmaxZ.first.get(),sizeof(pcl::PointXYZ),hipMemcpyDeviceToHost);
		if(err != ::hipSuccess)return err;
		err = hipMemcpy(&maxZ,minmaxZ.second.get(),sizeof(pcl::PointXYZ),hipMemcpyDeviceToHost);
		if(err != ::hipSuccess)return err;
		err = hipMemcpy(&minY,minmaxY.first.get(),sizeof(pcl::PointXYZ),hipMemcpyDeviceToHost);
		if(err != ::hipSuccess)return err;
		err = hipMemcpy(&maxY,minmaxY.second.get(),sizeof(pcl::PointXYZ),hipMemcpyDeviceToHost);
		if(err != ::hipSuccess)return err;
	
		int number_of_buckets_X=((maxX.x-minX.x)/resolution_X)+1;
		int number_of_buckets_Y=((maxY.y-minY.y)/resolution_Y)+1;
		int number_of_buckets_Z=((maxZ.z-minZ.z)/resolution_Z)+1;

		out_rgd_params.number_of_buckets_X=number_of_buckets_X;
		out_rgd_params.number_of_buckets_Y=number_of_buckets_Y;
		out_rgd_params.number_of_buckets_Z=number_of_buckets_Z;
		out_rgd_params.number_of_buckets = number_of_buckets_X * number_of_buckets_Y * number_of_buckets_Z;
		
		out_rgd_params.bounding_box_max_X=maxX.x;
		out_rgd_params.bounding_box_min_X=minX.x;
		out_rgd_params.bounding_box_max_Y=maxY.y;
		out_rgd_params.bounding_box_min_Y=minY.y;
		out_rgd_params.bounding_box_max_Z=maxZ.z;
		out_rgd_params.bounding_box_min_Z=minZ.z;
		
		out_rgd_params.resolution_X=resolution_X;
		out_rgd_params.resolution_Y=resolution_Y;
		out_rgd_params.resolution_Z=resolution_Z;
	}
	catch(thrust::system_error &e)
	{
		err = hipGetLastError();
		hipDeviceReset();
		return err;
	}	
	catch(std::bad_alloc &e)
  	{
  	 	err = hipGetLastError();
		hipDeviceReset();
		return err;
  	}	
	return hipGetLastError();
}

__global__ void kernel_initializeIndByKey(hashElement* d_hashTable, int number_of_points)
{
	int ind=blockIdx.x*blockDim.x+threadIdx.x;
	if(ind < number_of_points)
	{
		d_hashTable[ind].index_of_point=ind;
		d_hashTable[ind].index_of_bucket=0;
	}
}

__global__ void kernel_getIndexOfBucketForPoints(pcl::PointXYZ* cloud, hashElement* d_hashTable, int number_of_points, gridParameters rgd_params)
{
	int ind=blockIdx.x*blockDim.x+threadIdx.x;
	if(ind < number_of_points)
	{
		int ix=(cloud[ind].x-rgd_params.bounding_box_min_X)/rgd_params.resolution_X;
		int iy=(cloud[ind].y-rgd_params.bounding_box_min_Y)/rgd_params.resolution_Y;
		int iz=(cloud[ind].z-rgd_params.bounding_box_min_Z)/rgd_params.resolution_Z;
		d_hashTable[ind].index_of_bucket=ix*rgd_params.number_of_buckets_Y*rgd_params.number_of_buckets_Z+iy*rgd_params.number_of_buckets_Z+iz;
	}
}

__global__ void kernel_initializeBuckets(bucket* d_buckets, gridParameters rgd_params)
{
	long long int ind=blockIdx.x*blockDim.x+threadIdx.x;
	if(ind < rgd_params.number_of_buckets)
	{
		d_buckets[ind].index_begin=-1;
		d_buckets[ind].index_end=-1;
		d_buckets[ind].number_of_points=0;
	}
}

__global__ void kernel_updateBuckets(hashElement* d_hashTable, bucket* d_buckets,
		gridParameters rgd_params, int number_of_points)
{
	int ind = blockIdx.x*blockDim.x+threadIdx.x;
	if(ind < number_of_points)
	{
		if(ind == 0)
		{
			int index_of_bucket = d_hashTable[ind].index_of_bucket;
			int index_of_bucket_1 = d_hashTable[ind+1].index_of_bucket;

			d_buckets[index_of_bucket].index_begin=ind;
			if(index_of_bucket != index_of_bucket_1)
			{
				d_buckets[index_of_bucket].index_end=ind+1;
				d_buckets[index_of_bucket_1].index_end=ind+1;
			}
		}else if(ind == number_of_points-1)
		{
			d_buckets[d_hashTable[ind].index_of_bucket].index_end=ind+1;
		}else
		{
			int index_of_bucket = d_hashTable[ind].index_of_bucket;
			int index_of_bucket_1 = d_hashTable[ind+1].index_of_bucket;

			if(index_of_bucket != index_of_bucket_1)
			{
				d_buckets[index_of_bucket].index_end=ind+1;
				d_buckets[index_of_bucket_1].index_begin=ind+1;
			}
		}
	}
}

__global__ void kernel_countNumberOfPointsForBuckets(bucket* d_buckets, gridParameters rgd_params)
{
	int ind=blockIdx.x*blockDim.x+threadIdx.x;
	if(ind < rgd_params.number_of_buckets)
	{
		int index_begin = d_buckets[ind].index_begin;
		int index_end = d_buckets[ind].index_end;

		if(index_begin != -1 && index_end !=-1)
		{
			d_buckets[ind].number_of_points = index_end - index_begin;
		}
	}
}

__global__ void kernel_copyKeys(hashElement* d_hashTable_in, hashElement* d_hashTable_out, int number_of_points)
{
	int ind=blockIdx.x*blockDim.x+threadIdx.x;
	if(ind < number_of_points)
	{
		d_hashTable_out[ind] = d_hashTable_in[ind];
	}
}

hipError_t cudaCalculateGrid(int threads, pcl::PointXYZ *d_point_cloud, bucket *d_buckets,
		hashElement *d_hashTable, int number_of_points, gridParameters rgd_params)
{
	hipError_t err = hipGetLastError();
	hashElement* d_temp_hashTable;	hipMalloc((void**)&d_temp_hashTable,number_of_points*sizeof(hashElement));
	int blocks=number_of_points/threads + 1;
	
	kernel_initializeIndByKey<<<blocks,threads>>>(d_temp_hashTable, number_of_points);
	err = hipDeviceSynchronize();	if(err != ::hipSuccess)return err;
	
	kernel_getIndexOfBucketForPoints<<<blocks,threads>>>(d_point_cloud, d_temp_hashTable, number_of_points, rgd_params);
	err = hipDeviceSynchronize();	if(err != ::hipSuccess)return err;

	try
	{
		thrust::device_ptr<hashElement> t_d_temp_hashTable(d_temp_hashTable);
		thrust::sort(t_d_temp_hashTable,t_d_temp_hashTable+number_of_points,compareHashElements());
	}
	catch(thrust::system_error &e)	
	{
		err = hipGetLastError(); 
		return err;
	}	
	catch(std::bad_alloc &e)
	{
		err = hipGetLastError(); 
		return err;
	}	

	kernel_initializeBuckets<<<rgd_params.number_of_buckets/threads+1,threads>>>(d_buckets,rgd_params);
	err = hipDeviceSynchronize();	if(err != ::hipSuccess)return err;
	
	kernel_updateBuckets<<<blocks,threads>>>(d_temp_hashTable, d_buckets, rgd_params, number_of_points);
	err = hipDeviceSynchronize();	if(err != ::hipSuccess)return err;
	
	kernel_countNumberOfPointsForBuckets<<<rgd_params.number_of_buckets/threads+1,threads>>>(d_buckets, rgd_params);
	err = hipDeviceSynchronize();	if(err != ::hipSuccess)return err;
	
	kernel_copyKeys<<<blocks,threads>>>(d_temp_hashTable, d_hashTable, number_of_points);
	err = hipDeviceSynchronize(); if(err != ::hipSuccess)return err;
	
	err = hipFree(d_temp_hashTable);
	return err;
}

__global__ void kernel_setAllPointsToRemove(int number_of_points, bool *d_markers_out)
{
	int ind=blockIdx.x*blockDim.x+threadIdx.x;
	if(ind<number_of_points)
	{
		d_markers_out[ind] = false;
	}
}

__global__ void kernel_markPointsToRemain(pcl::PointXYZ *d_point_cloud,
		int number_of_points,
		hashElement *d_hashTable,
		bucket *d_buckets,
		gridParameters rgd_params,
		float search_radius,
		int number_of_points_in_search_sphere_threshold,
		int max_number_considered_in_INNER_bucket,
		int max_number_considered_in_OUTER_bucket,
		bool *d_markers_out)
{
	int index_of_point = blockIdx.x*blockDim.x+threadIdx.x;

	if(index_of_point < number_of_points)
	{
		int number_of_found_points_in_search_sphere_threshold = 0;

		float x = d_point_cloud[index_of_point].x;
		float y = d_point_cloud[index_of_point].y;
		float z = d_point_cloud[index_of_point].z;

		if(x < rgd_params.bounding_box_min_X || x > rgd_params.bounding_box_max_X)
		{
			d_markers_out[index_of_point] = false;
			return;
		}
		if(y < rgd_params.bounding_box_min_Y || y > rgd_params.bounding_box_max_Y)
		{
			d_markers_out[index_of_point] = false;
			return;
		}
		if(z < rgd_params.bounding_box_min_Z || z > rgd_params.bounding_box_max_Z)
		{
			d_markers_out[index_of_point] = false;
			return;
		}

		int ix=(x - rgd_params.bounding_box_min_X)/rgd_params.resolution_X;
		int iy=(y - rgd_params.bounding_box_min_Y)/rgd_params.resolution_Y;
		int iz=(z - rgd_params.bounding_box_min_Z)/rgd_params.resolution_Z;

		int index_bucket = ix*rgd_params.number_of_buckets_Y *
				rgd_params.number_of_buckets_Z + iy * rgd_params.number_of_buckets_Z + iz;


		if(index_bucket >= 0 && index_bucket < rgd_params.number_of_buckets)
		{
			int sx, sy, sz, stx, sty, stz;
			if(ix == 0)sx = 0; else sx = -1;
			if(iy == 0)sy = 0; else sy = -1;
			if(iz == 0)sz = 0; else sz =- 1;

			if(ix == rgd_params.number_of_buckets_X - 1)stx = 1; else stx = 2;
			if(iy == rgd_params.number_of_buckets_Y - 1)sty = 1; else sty = 2;
			if(iz == rgd_params.number_of_buckets_Z - 1)stz = 1; else stz = 2;

			int index_next_bucket;
			int iter;
			int number_of_points_in_bucket;
			int l_begin;
			int l_end;

			for(int i = sx; i < stx; i++)
			{
				for(int j = sy; j < sty; j++)
				{
					for(int k = sz; k < stz; k++)
					{
						index_next_bucket = index_bucket +
								i * rgd_params.number_of_buckets_Y * rgd_params.number_of_buckets_Z +
								j * rgd_params.number_of_buckets_Z + k;
						if(index_next_bucket >= 0 && index_next_bucket < rgd_params.number_of_buckets)
						{
							number_of_points_in_bucket = d_buckets[index_next_bucket].number_of_points;
							if(number_of_points_in_bucket <= 0)continue;

							int max_number_considered_in_bucket;
							if(index_next_bucket == index_bucket)
							{
								max_number_considered_in_bucket = max_number_considered_in_INNER_bucket;
							}else
							{
								max_number_considered_in_bucket = max_number_considered_in_OUTER_bucket;
							}
							if(max_number_considered_in_bucket <= 0)continue;

							if(max_number_considered_in_bucket >= number_of_points_in_bucket)
							{
								iter=1;
							}else
							{
								iter = number_of_points_in_bucket / max_number_considered_in_bucket;
								if(iter <= 0)iter = 1;
							}

							l_begin = d_buckets[index_next_bucket].index_begin;
							l_end = d_buckets[index_next_bucket].index_end;

							for(int l = l_begin; l < l_end; l += iter)
							{
								if(l >= 0 && l < number_of_points)
								{
									int hashed_index_of_point = d_hashTable[l].index_of_point;

									float nn_x  = d_point_cloud[hashed_index_of_point].x;
									float nn_y  = d_point_cloud[hashed_index_of_point].y;
									float nn_z  = d_point_cloud[hashed_index_of_point].z;

									float dist  = (x - nn_x) * (x - nn_x) +
												  (y - nn_y) * (y - nn_y) +
												  (z - nn_z) * (z - nn_z);

									if(dist <= search_radius * search_radius)
									{
										number_of_found_points_in_search_sphere_threshold++;
									}
								}
							}
						}
					}
				}
			}
		}

		if(number_of_found_points_in_search_sphere_threshold >= number_of_points_in_search_sphere_threshold + 1)
		{
			d_markers_out[index_of_point] = true;
		}else
		{
			d_markers_out[index_of_point] = false;
		}
	}
}

hipError_t cudaRemoveNoise(
			int threads,
			pcl::PointXYZ *d_point_cloud,
			int number_of_points,
			hashElement *d_hashTable,
			bucket *d_buckets,
			gridParameters rgd_params,
			float search_radius,
			int number_of_points_in_search_sphere_threshold,
			int max_number_considered_in_INNER_bucket,
			int max_number_considered_in_OUTER_bucket,
			bool *d_markers_out)
{
	hipError_t err = hipGetLastError();

	kernel_setAllPointsToRemove<<<number_of_points/threads+1,threads>>>(number_of_points, d_markers_out);
	err = hipDeviceSynchronize();	if(err != ::hipSuccess)return err;

	kernel_markPointsToRemain<<<number_of_points/threads+1,threads>>>
				   (d_point_cloud,
					number_of_points,
					d_hashTable,
					d_buckets,
					rgd_params,
					search_radius,
					number_of_points_in_search_sphere_threshold,
					max_number_considered_in_INNER_bucket,
					max_number_considered_in_OUTER_bucket,
					d_markers_out);
	err = hipDeviceSynchronize();

	return err;
}
